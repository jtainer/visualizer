// 
// DFT class implementation
// 
// 2022, Jonathan Tainer
// 

#include "cudaft.h"
#include "complex.h"
#include "matmul.h"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

CudaFT::CudaFT() {
	N = 0;
	devMatrix = NULL;
	devInput = NULL;
	devOutput = NULL;
}

CudaFT::~CudaFT() {
	hipFree(devMatrix);
	hipFree(devInput);
	hipFree(devOutput);
}

void CudaFT::setDims(unsigned int n) {
	
	// Allocate or reallocate an appropriate amount of GPU memory
	hipFree(devMatrix);
	hipFree(devInput);
	hipFree(devOutput);

	N = n;

	hipMalloc((void**)&devMatrix, sizeof(Complex) * N * N);
	hipMalloc((void**)&devInput, sizeof(float) * N);
	hipMalloc((void**)&devOutput, sizeof(Complex) * N);

	// Call kernel to construct DFT matrix in GPU memory
	genmat<<<(N * N / 256) + 1, 256>>>(devMatrix, N);

}

void CudaFT::transform(float* inputBuffer, Complex* outputBuffer) {
	
	// Copy input buffer to GPU memory
	hipMemcpy(devInput, inputBuffer, sizeof(float) * N, hipMemcpyHostToDevice);
	
	// Spawn kernel threads
	matmul<<<(N / 256) + 1, 256>>>(devMatrix, devInput, devOutput, N);
	
	// Copy output buffer from GPU memory to system memory
	hipMemcpy(outputBuffer, devOutput, sizeof(Complex) * N, hipMemcpyDeviceToHost);
}

void CudaFT::transformMag(float* inputBuffer, float* outputBuffer) {
	// Copy input buffer to GPU memory
	hipMemcpy(devInput, inputBuffer, sizeof(float) * N, hipMemcpyHostToDevice);

	// Spawn kernel threads
	matmulMag<<<(N / 256) + 1, 256>>>(devMatrix, devInput, (float*) devOutput, N);

	// Copy output buffer from GPU memory to system memory
	hipMemcpy(outputBuffer, devOutput, sizeof(float) * N, hipMemcpyDeviceToHost);
}















